#include "hip/hip_runtime.h"
#include "processing_gpu.cuh"
#include <>


__global__ void rdComplexMultiply(hipFloatComplex* s, hipFloatComplex* w, long int M, long int N)         //�����nmlgb������ѹ��, ���Ǹ�bƥ���˲����Ǹ�ɶ��
{
    long int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N * M)
    {
        long int n = i % N;
        printf("%d %d          ", i, n);
        s[i] = hipCmulf(s[i], hipConjf(w[n]));
    }
}


__global__ void rdComplexTranspose(hipFloatComplex* sout, hipFloatComplex* sin, long int M, long int N)       //����ת��???   �ǵ�
{
    long int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N * M)
    {
        long int n = i % N;                                                             //�������
        long int m = (long int)(i - n) / N;                                             //�������

        sout[m + n * M] = sin[n + m * N];
    }
 }


void readData(hipFloatComplex* signal)
{
    int a = 1;                              //��֪��զд     ���ص���Host����
}

 void pulseCompression(hipFloatComplex* d_signal, hipFloatComplex * d_ori, long int M, long int N)
 { 

    hipfftHandle plan1;
    hipfftPlan1d(&plan1, N, HIPFFT_C2C, M);
    hipfftExecC2C(plan1, (hipfftComplex*)d_signal, (hipfftComplex*)d_signal, HIPFFT_FORWARD);
    hipfftDestroy(plan1);

    hipfftHandle plan2;
    hipfftPlan1d(&plan2, N, HIPFFT_C2C, 1);
    hipfftExecC2C(plan2, (hipfftComplex*)d_ori, (hipfftComplex*)d_ori, HIPFFT_FORWARD);
    hipfftDestroy(plan2);
    
    dim3 block, grid;
    block.x = 1024;
    grid.x = (M * N + block.x - 1) / block.x;
    rdComplexMultiply<<<block,grid>>>(d_signal, d_ori, M, N);                               //��ͻȻ����,����û�ù���, nvprof��ɶnightSystemɶ��Ҳ��û������
    
    hipfftHandle plan3;
    hipfftPlan1d(&plan3, N, HIPFFT_C2C, M);
    hipfftExecC2C(plan3, d_signal, d_signal, HIPFFT_BACKWARD);
    hipfftDestroy(plan3);

 }

 void mtd(hipFloatComplex* d_signal, long int M, long int N)
 {
     /*hipFloatComplex* signal;
     size_t memSize = M* N * sizeof(hipFloatComplex);
     hipMalloc((void**)&signal, memSize);
     hipMemcpy(signal, d_signal, memSize, hipMemcpyDeviceToDevice);               //����, ���滹��ת�û�ȥ, cfarҪ��mtd�Ľ��*/
     dim3 block, grid;
     block.x = 1024;
     grid.x = (M * N + block.x - 1) / block.x;
     rdComplexTranspose << <block, grid >> > (d_signal, d_signal, M, N);                                  //����úݺݵ��Ż�
     hipfftHandle plan;
     hipfftPlan1d(&plan,M,HIPFFT_C2C,N);
     hipfftExecC2C(plan, d_signal, d_signal, HIPFFT_FORWARD);
     hipfftDestroy(plan);
     rdComplexTranspose << <block, grid >> > (d_signal, d_signal, M, N);
 }

 void CFAR(hipFloatComplex* d_signal, long int M, long int N, int rnum, int prum)                                  //���ȡԪ��[i-pnum+rnum:ii-pnum-1 i+pnum+1:ii+rnum+pnum]��û���Ż�������, ���������ʲô�Ķ���ûŪ
 {
     long int i = blockIdx.x* blockDim.x + threadIdx.x;         //���զŪaaaaa
     if(i>)
 }
 void doGpuProcessing(hipFloatComplex* signal, hipFloatComplex* ori, long int M, long int N)
{
     size_t memSize = M * N * sizeof(hipFloatComplex);
     hipFloatComplex* d_signal, * d_ori;
     hipMalloc((void**)&d_signal, memSize);
     hipMalloc((void**)&d_ori, memSize / M);
     hipMemcpy(d_signal, signal, memSize, hipMemcpyHostToDevice);
     hipMemcpy(d_ori, ori, memSize, hipMemcpyHostToDevice);
     pulseCompression(d_signal, d_ori, M, N);
     mtd(d_signal, M, N);

}