#include "hip/hip_runtime.h"
#include "processing_gpu.cuh"
#include <>


__global__ void rdComplexMultiply(hipFloatComplex* s, hipFloatComplex* w, long int M, long int N)         //�����nmlgb������ѹ��, ���Ǹ�bƥ���˲����Ǹ�ɶ��
{
    long int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N * M)
    {
        long int n = i % N;
        printf("%d %d          ", i, n);
        s[i] = hipCmulf(s[i], hipConjf(w[n]));
    }
}


void readData(hipFloatComplex* signal)
{
    int a = 1;                              //��֪��զд
}

 void pulseCompression(hipFloatComplex* signal, hipFloatComplex *ori, int M, int N)
{
    size_t memSize = M * N * sizeof(hipFloatComplex);
    hipFloatComplex* d_signal, * d_ori;
    hipMalloc((void**)&d_signal, memSize);
    hipMalloc((void**)&d_ori, memSize / M);
    hipMemcpy(d_signal, signal, memSize, hipMemcpyHostToDevice);
    hipMemcpy(d_ori, ori, memSize, hipMemcpyHostToDevice);

    hipfftHandle plan1;
    hipfftPlan1d(&plan1, N, HIPFFT_C2C, M);
    hipfftExecC2C(plan1, (hipfftComplex*)d_signal, (hipfftComplex*)d_signal, HIPFFT_FORWARD);
    hipfftDestroy(plan1);

    hipfftHandle plan2;
    hipfftPlan1d(&plan2, N, HIPFFT_C2C, 1);
    hipfftExecC2C(plan2, (hipfftComplex*)d_ori, (hipfftComplex*)d_ori, HIPFFT_FORWARD);
    hipfftDestroy(plan2);
    
    dim3 block, grid;
    block.x = 1024;
    grid.x = (M * N + block.x - 1) / block.x;
    rdComplexMultiply<<<block,grid>>>(d_signal, d_ori, M, N);                               //��ͻȻ����,����û�ù���, nvprof��ɶnightSystemɶ��Ҳ��û������
    
    hipfftHandle plan3;
    hipfftPlan1d(&plan3, M, HIPFFT_C2C, N);
    hipfftExecC2C(plan3, d_signal, d_signal, HIPFFT_BACKWARD);
    hipfftDestroy(plan3);
}