#include "hip/hip_runtime.h"
#include "processing_gpu.cuh"
#include <>

//#include "matplotlibcpp.h"
//namespace plt = matplotlibcpp;

#define BLOCKX 256

void dev_setup(int M,int N)
{
    // set up device
    int dev = 0, driverVersion = 0, runtimeVersion = 0;
    hipSetDevice(dev);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);

    // check if support mapped memory
    if (!deviceProp.canMapHostMemory)
    {
        printf("Device %d does not support mapping CPU host memory!\n", dev);
        hipDeviceReset();
        exit(EXIT_SUCCESS);
    }

    printf("  GPU�豸����                         %s \n", deviceProp.name);
    printf("  GPU������������SM������             %d \n", (int)deviceProp.multiProcessorCount);
    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);
    printf("  CUDA Driver �汾 / Runtime �汾     %d.%d / %d.%d\n",
        driverVersion / 1000, (driverVersion % 100) / 10,
        runtimeVersion / 1000, (runtimeVersion % 100) / 10);
    printf("  CUDA ������:                       %d.%d\n",
        deviceProp.major, deviceProp.minor);
    printf("  �Դ��С:                          %.2f GBytes (%llu "
        "bytes)\n", (float)deviceProp.totalGlobalMem / pow(1024.0, 3),
        (unsigned long long)deviceProp.totalGlobalMem);
    printf("  GPU ʱ��Ƶ��:                      %.0f MHz (%0.2f "
        "GHz)\n", deviceProp.clockRate * 1e-3f,
        deviceProp.clockRate * 1e-6f);
    printf("  Memory ʱ��Ƶ��:                   %.0f Mhz\n",
        deviceProp.memoryClockRate * 1e-3f);
    printf("  �����ģ��                         %d * %d\n", M, N);
}

__global__ void rdComplexMultiply(hipDoubleComplex* s, hipDoubleComplex* w, int M, int N)         //�����nmlgb������ѹ��, ���Ǹ�bƥ���˲����Ǹ�ɶ��
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N * M)
    {
        int n = i % N;
        //printf("%d %d          ", i, n);
        s[i] = hipCmul(s[i], hipConj(w[n]));
    }
}


__global__ void rdComplexTranspose(hipDoubleComplex* sout, hipDoubleComplex* sin, int M, int N)       //����ת��???   �ǵ�      ��������nmdgb bug, sin��sout������ͬһ����Ȼ���ͻ
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N * M)
    {
        int n = i % N;                                                             //�������
        int m = (int)(i - n) / N;                                             //�������

        sout[m + n * M] = sin[n + m * N];
    }
 }


/*__global__ void rdSquareCopy(hipDoubleComplex* sout, hipDoubleComplex* sin, int M, int N) {           //sinƽ��Ȼ�������cout��   //��˵�㶼ƽ����, ��������ɶ, ����!?
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N * M)
    {
        double x = hipCabsf(sin[i]);                                          //��ģ
        sout[i] = make_hipFloatComplex(x * x, 0);                            //ƽ��, ת����
    }
}*/


__global__ void rdSquareCopy(double* sout, hipDoubleComplex* sin, int M, int N) {           //sinƽ��Ȼ�������cout��   //��˵�㶼ƽ����, ��������ɶ, ����!?
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N * M)
    {
        double x = hipCabs(sin[i]);                                          //��ģ
        sout[i] = x*x;                                               //ƽ��, ת����
    }
    
}



void readData(hipDoubleComplex* signal, hipDoubleComplex *ori, int M, int N)                    //�����ʱ���ù̶��ڴ��ܺõ�
{
    FILE* fp;//�ļ�ָ��
    fp = fopen("signal_real.txt", "r");//���ı���ʽ���ļ���
    if (fp == NULL) //���ļ�����
        printf("error1");
    for (int i = 0; i<M*N; i++)
    {
        fscanf(fp, "%lf", &signal[i].x);
  
    }
    fclose(fp);//�ر��ļ�
    fp = fopen("signal_imag.txt", "r");
    if (fp == NULL)
        printf("error");
    for (int i = 0; i < M * N; i++)
    {
        fscanf(fp, "%lf", &signal[i].y);

    }
    fclose(fp);
    fp = fopen("ori_real.txt", "r");
    if (fp == NULL)
        printf("error");
    for (int i = 0; i<N; i++)
    {
        fscanf(fp, "%lf", &ori[i].x);

    }
    fclose(fp);
    fp = fopen("ori_imag.txt", "r");
    if (fp == NULL)
        printf("error");
    for (int i = 0; i < N; i++)
    {
        fscanf(fp, "%lf", &ori[i].y);
    }
    fclose(fp);
    printf("������, �����! \n");
}

void writeDataComplex(hipDoubleComplex* d_signal, int M, int N)                    
{
    int memSize = M * N * sizeof(hipDoubleComplex);
    hipDoubleComplex* signal;
    hipHostMalloc((void**)&signal,memSize);
    hipMemcpy(signal, d_signal, memSize, hipMemcpyDeviceToHost);
    //hipMemset(signal, 1145, memSize);
    test(d_signal, M, N);
    FILE* fp;//�ļ�ָ��
    fp = fopen("signal_real_out.txt", "w");//���ı���ʽ���ļ���
    printf("caonima, %lf", signal[6].y);
    if (fp == NULL) //���ļ�����
        printf("error1");
    for (int i = 0; i < M * N; i++)
    {
        fprintf(fp, "%lf\n", signal[i].x);

    }
    fclose(fp);//�ر��ļ�
    fp = fopen("signal_imag_out.txt", "w");
    if (fp == NULL)
        printf("error");
    for (int i = 0; i < M * N; i++)
    {
        fprintf(fp, "%lf\n", signal[i].y);
        

    }
    fclose(fp);
    
    printf("д����, �����! \n");
}

void writeData (double *d_signal, int M, int N)               //����������gpu���ڴ������, �����Զ����㿽����Host��
{
    size_t memSize = M * N * sizeof(double);
    double* out;
    hipHostMalloc((void**)&out, memSize);
    hipMemcpy(out, d_signal, memSize, hipMemcpyDeviceToHost);
    //out[1] = 1;
    FILE* fpWrite;
    fpWrite = fopen("writeData_out.txt", "w");
    if (fpWrite == NULL)
    {
        printf("error");
        return;
    }
    for (int i = 0; i < M * N; i++)
        fprintf(fpWrite, "%2.15f\n", out[i]);
    fclose(fpWrite);
}

/*����ѹ��
* d_signal�ǻز�, d_ori��sin�ź�, M, N���źŵ�M��N��
*/

 void pulseCompression(hipDoubleComplex* d_signal, hipDoubleComplex * d_ori, int M, int N)                
 { 
    int inembed[1] = { 0 };
    int onembed[1] = { 0 };
    int number_N[1] = { (int)N };                                         //�ⲻ��longint��ͻ��
    int istride = 1;
    int rank = 1;
    int ostride = 1;
    hipfftHandle plan1;
    
    //hipfftPlanMany(&plan1, rank, number_N, inembed, istride, N, onembed, ostride, N, HIPFFT_Z2Z, M);
    hipfftPlan1d(&plan1, N, HIPFFT_Z2Z, M);
    hipfftExecZ2Z(plan1, (hipfftDoubleComplex*)d_signal, (hipfftDoubleComplex*)d_signal, HIPFFT_FORWARD);               //�ź�fft, �������
    //printf("fuck ");
    //printGpuModComplex(d_signal);
    hipfftDestroy(plan1);
    //test(d_signal, M, N);
    hipfftHandle plan2;
    hipfftPlan1d(&plan2, N, HIPFFT_Z2Z, 1);
    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)d_ori, (hipfftDoubleComplex*)d_ori, HIPFFT_FORWARD);                     //sin fft, �������
    hipfftDestroy(plan2);
    
    dim3 block, grid;
    block.x = BLOCKX;
    grid.x = (M * N + block.x - 1) / block.x;
    //printf("%d, %d \n", block.x, grid.x);
    rdComplexMultiply<<<grid, block >>>(d_signal, d_ori, M, N);                                              //���Թ���ֱ, ֱ�Ӹĵ�d_signal
    //test(d_ori, 1, N);                                                                                                                                               //��ͻȻ����,����û�ù���, nvprof��ɶnightSystemɶ��Ҳ��û������
    
    hipfftHandle plan3;
    hipfftPlan1d(&plan3, N, HIPFFT_Z2Z, M);
    hipfftExecZ2Z(plan3, (hipfftDoubleComplex*)d_signal, (hipfftDoubleComplex*)d_signal, HIPFFT_BACKWARD);                                                 //ifft
    hipfftDestroy(plan3);
 }

 /*mtd
* d_signal�ǻز�, d_ori��sin�ź�, M, N���źŵ�M��N��
*/

 void mtd(hipDoubleComplex* d_signal, int M, int N)
 {
     /*hipDoubleComplex* signal;
     size_t memSize = M* N * sizeof(hipDoubleComplex);
     hipMalloc((void**)&signal, memSize);  
     hipMemcpy(signal, d_signal, memSize, hipMemcpyDeviceToDevice);                                                                         //����, ���滹��ת�û�ȥ, cfarҪ��mtd�Ľ��*/
     dim3 block, grid;
     block.x = BLOCKX;
     grid.x = (M * N + block.x - 1) / block.x;
     hipDoubleComplex* dd_signal;
     size_t memSize = M * N * sizeof(hipDoubleComplex);
     hipMalloc((void**)&dd_signal, memSize);
     rdComplexTranspose <<< grid, block >>> (dd_signal, d_signal, M, N);                                    //��ת�ú����е�fft           //����úݺݵ��Ż�
     //writeDataComplex(dd_signal, M, N);
     hipfftHandle plan;
     hipfftPlan1d(&plan,M,HIPFFT_Z2Z,N);                                                                                                          //����˵fft����Ӧ�ô���M���� ��һ��k>M
     hipfftExecZ2Z(plan, (hipfftDoubleComplex*)dd_signal, (hipfftDoubleComplex*)dd_signal, HIPFFT_FORWARD);                                                 //��fft
     hipfftDestroy(plan);
     rdComplexTranspose <<< grid, block >>> (d_signal, dd_signal, N, M);                                 //ת�û�ȥ��ʱ����N��M��, ������N,M!!!!!
     hipFree(dd_signal);
 }


 /*CFAR
* d_signal�ǻز�, d_ori��sin�ź�, M, N���źŵ�M��N��, rnum�ǲο���Ԫ����, pnum�Ǳ�����Ԫ����, k���ĸ����ڹ���ƽ��ֵ�ϵ��Ǹ�ϵ��(����pfa�����˵��Ǹ�)
*/

 __global__ void CFAR(double *d_out, double* d_signal, int M, int N ,int rnum, int pnum, double k)                                                  // ���������ʲô�Ķ���ûŪ
 {
     int i = blockIdx.x* blockDim.x + threadIdx.x;                                                                                                            //���զŪaaaaa
     int col = i % N;                                                                  //i% N��������, ��ÿһ�еĵڼ���Ԫ��
     double thold=0;
     if (col >= rnum + pnum && col < N - rnum - pnum && i < M * N)                          //��Ե����û�� ���ȡԪ��[i-pnum+rnum:ii-pnum-1 i+pnum+1:ii+rnum+pnum]��û���Ż�������
     {                                                                                        //����ж��źŴ󲻴������޵ĺ����о�����һ�ѷ�֧ɶ��
         for (int aaa = pnum + 1; aaa <=pnum + rnum; aaa++)                                                     //զ�Ż�, �о��ǲ��ǵû���������������
         {
             d_out[i] = d_out[i] + d_signal[i + aaa] + d_signal[i - aaa];
         }
         d_out[i] = d_out[i] / (double)rnum;
         thold = d_out[i] * k;
         if(d_signal[i]<= thold)
         { 
             d_out[i] = 0;                                                                      //���Ƿ�������, �������޵ı���, С�ڵ�����    //�о���Ҳ���Ż��ռ�
         }
     }
 }
 /*
 void test(hipDoubleComplex* d_signal, int M, int N)
 {
     size_t memSize;
     memSize = M * N * sizeof(double);
     double* a;
     hipMalloc((void**)&a, memSize);
     hipMemset(a, 0, memSize);
     dim3 blockkk, griddd;
     blockkk.x = 1024;
     griddd.x = (M * N + blockkk.x - 1) / blockkk.x;
     rdSquareCopy << < griddd, blockkk >> > (a, d_signal, M, N);
     printf("test out: ");
     printGpuModFloat(a);
     writeData(a, M, N);
     hipFree(a);
 }

 void printGpuModComplex(hipDoubleComplex  *d_signal)
 {
     hipDoubleComplex* a;
     hipHostMalloc((void**)&a, sizeof(hipDoubleComplex));
     hipMemcpy(a, d_signal, sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
     printf("%f   \n", hipCabs(a[0]));
     hipFree(a);
 }
 
 void printGpuModFloat(double* d_signal)
 {
     double* a;
     hipHostMalloc((void**)&a, sizeof(double));
     hipMemcpy(a, d_signal, sizeof(double), hipMemcpyDeviceToHost);
     printf("%f\n", a[0]);
 }

 void makeSmall(hipDoubleComplex* d_signal, int M, int N)
 {
     dim3 block, grid;
     block.x = BLOCKX;
     grid.x = (M * N + block.x - 1) / block.x;
     mod1w << <grid, block >> > (d_signal, M, N);
 }


 __global__ void mod1w(hipDoubleComplex* d_signal, int M, int N)
 {
     int i = blockIdx.x * blockDim.x + threadIdx.x;
     if (i < M * N)
     {
         d_signal[i] = make_hipDoubleComplex(hipCreal(d_signal[i]) / 100000, hipCimag(d_signal[i]) / 100000);
     }
 }
 */
 LARGE_INTEGER nFreq;
 LARGE_INTEGER nLastTime1;
 LARGE_INTEGER nLastTime2;
 static int __count = 0;
 float doGpuProcessing(hipDoubleComplex* signal, hipDoubleComplex* ori, int M, int N)
{
     //printf("%d ", ++__count);
     QueryPerformanceFrequency(&nFreq);
     QueryPerformanceCounter(&nLastTime1);
     size_t memSize = M * N * sizeof(hipDoubleComplex);
     hipDoubleComplex* d_signal, * d_ori;
     hipMalloc((void**)&d_signal, memSize);
     hipMalloc((void**)&d_ori, memSize / M);
     hipMemcpy(d_signal, signal, memSize, hipMemcpyHostToDevice);
     hipMemcpy(d_ori, ori, memSize/M, hipMemcpyHostToDevice);                           //signal��oriŪ��gpu��
     //test(d_signal, M, N);
     pulseCompression(d_signal, d_ori, M, N);                                           //����ѹ��
     //writeDataComplex(d_signal, M, N);
     //makeSmall(d_signal, M, N);
     //hipDeviceSynchronize();
     //test(d_signal, M, N);                                                            
     //makeSmall(d_signal, M, N);
     mtd(d_signal, M, N);                                                               //����ѹ���Ľ���͵�mtd
     //test(d_signal, M, N);
     double* d_sqSignal;
     hipMalloc((void**)&d_sqSignal, memSize);
     double* d_out;
     hipMalloc((void**)&d_out, memSize);
     hipMemset(d_out, 1, memSize);                                 //��Ҳ��֪����û��������һ��         �е�, ��Ȼ��Ե�ľ�û��ֵ��
     dim3 block1, grid1;
     block1.x = BLOCKX;
     grid1.x = (M * N + block1.x - 1) / block1.x;
     rdSquareCopy << < grid1, block1 >> > (d_sqSignal, d_signal, M, N);
     dim3 block2, grid2;
     block2.x = BLOCKX;
     grid2.x = (M * N + block2.x - 1) / block2.x;
     int pnum = 4;                                  //������Ԫ
     int rnum = 10;                                  // �ο���Ԫ
     double pfa = 1e-6;                                 // ���龯��               //������Կ������Ǹ�ʲôʲô�����ڴ�ɶ��
     double k = powf(pfa, (-1 / (2 * (double)rnum))) - 1;
     CFAR << < grid2, block2 >> > (d_out, d_sqSignal, M, N, rnum, pnum, k);
     QueryPerformanceCounter(&nLastTime2);
     float fInterval = nLastTime2.QuadPart - nLastTime1.QuadPart;
    
     //writeData(d_out, M, N);
     //printf("��������, ����� \n");
     hipFree(d_signal);
     hipFree(d_ori);
     hipFree(d_sqSignal);
     hipFree(d_out);
     return  fInterval / (float)nFreq.QuadPart;
}