﻿#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <>
#include <hipfft/hipfft.h>
#include <math.h>
#include <hip/hip_complex.h>
#include "hip/hip_runtime.h"
#include ""
#include "processing_gpu.cuh"
//#include "matplotlibcpp.h"
//namespace plt = matplotlibcpp;
int main()
{
																			
	int M = 64, N = 4000;																//M行 N列
	dev_setup(M, N);
	size_t memSize = M * N * sizeof(hipDoubleComplex);
	hipDoubleComplex* signal, * ori;
	hipHostMalloc((void**)&signal, memSize);												//用固定内存能快点, 在cpy的时候
	hipHostMalloc((void**)&ori, memSize/M);
	readData(signal, ori, M, N);															//读matlab的回波信号, 然后直接传给doGpuProcessing, 在那里面hipMalloc
	//printf("%lf", hipCreal(ori[1]));
	float time = 0;
	int runNum = 1000;
	for (int i = 0; i < runNum; i++)
	{
		time+=doGpuProcessing(signal, ori, M, N);									
	}
	time = time / runNum;
	hipFree(signal);
	hipFree(ori);
	printf("\n\naverage run time is: %.6f   \n", time);
	return 0;
}

