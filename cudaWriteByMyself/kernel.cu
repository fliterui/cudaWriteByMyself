﻿#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <>
#include <hipfft/hipfft.h>
#include <math.h>
#include <hip/hip_complex.h>
#include "hip/hip_runtime.h"
#include ""
#include "processing_gpu.cuh"
//#include "matplotlibcpp.h"
//namespace plt = matplotlibcpp;
int main()
{
	int M = 64, N = 4000;																//M行 N列
	size_t memSize = M * N * sizeof(hipDoubleComplex);
	hipDoubleComplex* signal, * ori;
	hipHostMalloc((void**)&signal, memSize);												//用固定内存能快点, 在cpy的时候
	hipHostMalloc((void**)&ori, memSize/M);
	readData(signal, ori, M, N);															//读matlab的回波信号, 然后直接传给doGpuProcessing, 在那里面hipMalloc
	//printf("%lf", hipCreal(ori[1]));
	doGpuProcessing(signal, ori, M, N);
	return 0;
}