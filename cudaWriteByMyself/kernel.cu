﻿#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <>
#include <hipfft/hipfft.h>
#include <math.h>
#include <hip/hip_complex.h>
#include "hip/hip_runtime.h"
#include ""
#include "processing_gpu.cuh"
#include "matplotlibcpp.h"
namespace plt = matplotlibcpp;
int main()
{
	/*long int M = 64, N = 4000;																//M行 N列
	size_t memSize = M * N * sizeof(hipFloatComplex);
	hipFloatComplex* signal, * ori;
	hipHostMalloc((void**)&signal, memSize);												//用固定内存能快点, 在cpy的时候
	hipHostMalloc((void**)&ori, memSize/M);
	readData(signal, ori, M, N);
	doGpuProcessing(signal, ori, M, N);*/
	plt::plot({ 1,2,3,4,5 });
	plt::show();
	return 0;
}