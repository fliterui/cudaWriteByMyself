#include "hip/hip_runtime.h"
#include"better_pack_fw.cuh"
#include"processing_gpu.cuh"

__global__ void better_rdComplexMultiply(hipDoubleComplex* s, hipDoubleComplex* w, int M, int N)         //Ϊ�˸���ת��Ū��
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N * M)
    {
        int n = i % N;
        //printf("%d %d          ", i, n);
        s[i] = hipCmul(s[i], hipConj(w[n]));
    }
}

__global__ void better_rdComplexTranspose(hipDoubleComplex* sout, hipDoubleComplex* sin, int M, int N)       //����ת��???   �ǵ�      ��������nmdgb bug, sin��sout������ͬһ����Ȼ���ͻ
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N * M)
    {
        int n = i % N;                                                             //�������
        int m = (int)(i - n) / N;                                             //�������

        sout[m + n * M] = sin[n + m * N];
    }
}

void better_mtd(hipDoubleComplex* d_signal, int M, int N, dim3 grid1, dim3 block1, dim3 grid2, dim3 block2)
{
    /*hipDoubleComplex* signal;
    size_t memSize = M* N * sizeof(hipDoubleComplex);
    hipMalloc((void**)&signal, memSize);
    hipMemcpy(signal, d_signal, memSize, hipMemcpyDeviceToDevice);                                                                         //����, ���滹��ת�û�ȥ, cfarҪ��mtd�Ľ��*/
    // block, grid;
    //block.x = BLOCKX;
    //grid.x = (M * N + block.x - 1) / block.x;
    hipDoubleComplex* dd_signal;
    size_t memSize = M * N * sizeof(hipDoubleComplex);
    hipMalloc((void**)&dd_signal, memSize);
    better_rdComplexTranspose << < grid1, block1 >> > (dd_signal, d_signal, M, N);                                    //��ת�ú����е�fft           //����úݺݵ��Ż�
    hipfftHandle plan;
    hipfftPlan1d(&plan, M, HIPFFT_Z2Z, N);                                                                                                          //����˵fft����Ӧ�ô���M���� ��һ��k>M
    hipfftExecZ2Z(plan, (hipfftDoubleComplex*)dd_signal, (hipfftDoubleComplex*)dd_signal, HIPFFT_FORWARD);                                                 //��fft
    hipfftDestroy(plan);
    rdComplexTranspose << < grid2, block2 >> > (d_signal, dd_signal, N, M);                 //ת�û�ȥ��ʱ����N��M��, ������N,M!!!!!   ������Ҳ�֪��զŪ,�ȴպ��ð�
    hipFree(dd_signal);
}
LARGE_INTEGER nFreq_1;
LARGE_INTEGER nLastTime1_1;
LARGE_INTEGER nLastTime2_1;
float goodTransDoGpuProcessing(hipDoubleComplex* signal, hipDoubleComplex* ori, int M, int N, dim3* grid, dim3* block)
{

    //printf("%d ", ++__count);
    QueryPerformanceFrequency(&nFreq_1);
    QueryPerformanceCounter(&nLastTime1_1);
    size_t memSize = M * N * sizeof(hipDoubleComplex);
    hipDoubleComplex* d_signal, * d_ori;
    hipMalloc((void**)&d_signal, memSize);
    hipMalloc((void**)&d_ori, memSize / M);
    //eee                     //signal��oriŪ��gpu��
    ///////////////////////////////Ϊ�˷��������ѹ���ĺ�������////////////////
    //pulseCompression(d_signal, d_ori, M, N, grid[0], block[0]);                                           //����ѹ��

    hipStream_t ps1;
    hipStream_t ps2;
    hipStreamCreateWithFlags(&ps1, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&ps2, hipStreamNonBlocking);
    hipMemcpyAsync(d_signal, signal, memSize, hipMemcpyHostToDevice, ps1);
    hipMemcpyAsync(d_ori, ori, memSize / M, hipMemcpyHostToDevice, ps2);

    hipfftHandle plan1;
    hipfftSetStream(plan1, ps1);
    hipfftHandle plan2;
    hipfftSetStream(plan2, ps2);
    hipfftPlan1d(&plan1, N, HIPFFT_Z2Z, M);
    hipfftExecZ2Z(plan1, (hipfftDoubleComplex*)d_signal, (hipfftDoubleComplex*)d_signal, HIPFFT_FORWARD);               //�ź�fft, �������
    hipfftDestroy(plan1);

    hipfftPlan1d(&plan2, N, HIPFFT_Z2Z, 1);
    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)d_ori, (hipfftDoubleComplex*)d_ori, HIPFFT_FORWARD);                     //sin fft, �������
    hipfftDestroy(plan2);
    hipStreamDestroy(ps1);
    hipStreamDestroy(ps2);
    rdComplexMultiply << <grid[0], block[0] >> > (d_signal, d_ori, M, N);                                              //���Թ���ֱ, ֱ�Ӹĵ�d_signal
    hipfftHandle plan3;
    hipfftPlan1d(&plan3, N, HIPFFT_Z2Z, M);
    hipfftExecZ2Z(plan3, (hipfftDoubleComplex*)d_signal, (hipfftDoubleComplex*)d_signal, HIPFFT_BACKWARD);                                                 //ifft
    hipfftDestroy(plan3);
    ////////////////////////////////////////////////////////////////////////////////
    better_mtd(d_signal, M, N, grid[1], block[1], grid[2], block[2]);                                                               //����ѹ���Ľ���͵�mtd
    //test(d_signal, M, N);
    double* d_sqSignal;
    hipMalloc((void**)&d_sqSignal, memSize);
    double* d_out;
    hipMalloc((void**)&d_out, memSize);
    hipMemset(d_out, 1, memSize);                                 //��Ҳ��֪����û��������һ��         �е�, ��Ȼ��Ե�ľ�û��ֵ��
    //dim3 block1, grid1;
    //block1.x = BLOCKX;
    //grid1.x = (M * N + block1.x - 1) / block1.x;
    rdSquareCopy << < grid[3], block[3] >> > (d_sqSignal, d_signal, M, N);
    //dim3 block2, grid2;
    //block2.x = BLOCKX;
    //grid2.x = (M * N + block2.x - 1) / block2.x;
    int pnum = 4;                                  //������Ԫ
    int rnum = 10;                                  // �ο���Ԫ
    double pfa = 1e-6;                                 // ���龯��               //������Կ������Ǹ�ʲôʲô�����ڴ�ɶ��
    double k = powf(pfa, (-1 / (2 * (double)rnum))) - 1;
    CFAR << < grid[4], block[4] >> > (d_out, d_sqSignal, M, N, rnum, pnum, k);
    QueryPerformanceCounter(&nLastTime2_1);
    float fInterval = nLastTime2_1.QuadPart - nLastTime1_1.QuadPart;

    //writeData(d_out, M, N);
    //printf("��������, ����� \n");
    hipFree(d_signal);
    hipFree(d_ori);
    hipFree(d_sqSignal);
    hipFree(d_out);
    return  fInterval / (float)nFreq_1.QuadPart;
}