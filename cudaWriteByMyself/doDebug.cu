#include "hip/hip_runtime.h"
#include "doDebug.cuh"
#include"processing_gpu.cuh"
#include <>

void test(hipDoubleComplex* d_signal, int M, int N)
{
    size_t memSize;
    memSize = M * N * sizeof(double);
    double* a;
    hipMalloc((void**)&a, memSize);
    hipMemset(a, 0, memSize);
    dim3 blockkk, griddd;
    blockkk.x = 1024;
    griddd.x = (M * N + blockkk.x - 1) / blockkk.x;
    rdSquareCopy << < griddd, blockkk >> > (a, d_signal, M, N);
    printf("test out: ");
    printGpuModFloat(a);
    writeData(a, M, N);
    hipFree(a);
}

void printGpuModComplex(hipDoubleComplex* d_signal)
{
    hipDoubleComplex* a;
    hipHostMalloc((void**)&a, sizeof(hipDoubleComplex));
    hipMemcpy(a, d_signal, sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    printf("%f   \n", hipCabs(a[0]));
    hipFree(a);
}

void printGpuModFloat(double* d_signal)
{
    double* a;
    hipHostMalloc((void**)&a, sizeof(double));
    hipMemcpy(a, d_signal, sizeof(double), hipMemcpyDeviceToHost);
    printf("%f\n", a[0]);
}

void makeSmall(hipDoubleComplex* d_signal, int M, int N)
{
    dim3 block, grid;
    block.x = 256;
    grid.x = (M * N + block.x - 1) / block.x;
    mod1w << <grid, block >> > (d_signal, M, N);
}


__global__ void mod1w(hipDoubleComplex* d_signal, int M, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < M * N)
    {
        d_signal[i] = make_hipDoubleComplex(hipCreal(d_signal[i]) / 100000, hipCimag(d_signal[i]) / 100000);
    }
}